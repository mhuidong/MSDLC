#include "hip/hip_runtime.h"
// Copyright (c) NXAI GmbH and its affiliates 2023
// Korbinian Poeppel
// Adapted from the haste library
//
// See:
// Copyright 2020 LMNT, Inc. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ==============================================================================

#include "blas.h"
#include "inline_print.cuh"

hipblasStatus_t cublasHgemmsb(hipblasHandle_t handle, hipblasOperation_t transa,
                             hipblasOperation_t transb, int m, int n, int k,
                             const __half *alpha, const __half *A, int lda,
                             long long int strideA, const __half *B, int ldb,
                             long long int strideB, const __half *beta,
                             __half *C, int ldc, long long int strideC,
                             int batchCount) {
  hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  hipDataType Atype = HIP_R_16F;
  hipDataType Btype = HIP_R_16F;
  hipDataType Ctype = HIP_R_16F;
  hipblasComputeType_t computeType = HIPBLAS_COMPUTE_16F;
  return hipblasGemmStridedBatchedEx(handle, transa, transb, m, n, k, alpha, A,
                                    Atype, lda, strideA, B, Btype, ldb, strideB,
                                    beta, C, Ctype, ldc, strideC, batchCount,
                                    computeType, algo);
}

hipblasStatus_t cublasSgemmsb(hipblasHandle_t handle, hipblasOperation_t transa,
                             hipblasOperation_t transb, int m, int n, int k,
                             const float *alpha, const float *A, int lda,
                             long long int strideA, const float *B, int ldb,
                             long long int strideB, const float *beta, float *C,
                             int ldc, long long int strideC, int batchCount) {
  hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  hipDataType Atype = HIP_R_32F;
  hipDataType Btype = HIP_R_32F;
  hipDataType Ctype = HIP_R_32F;
  hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32F;
  return hipblasGemmStridedBatchedEx(handle, transa, transb, m, n, k, alpha, A,
                                    Atype, lda, strideA, B, Btype, ldb, strideB,
                                    beta, C, Ctype, ldc, strideC, batchCount,
                                    computeType, algo);
}

hipblasStatus_t cublasDgemmsb(hipblasHandle_t handle, hipblasOperation_t transa,
                             hipblasOperation_t transb, int m, int n, int k,
                             const double *alpha, const double *A, int lda,
                             long long int strideA, const double *B, int ldb,
                             long long int strideB, const double *beta,
                             double *C, int ldc, long long int strideC,
                             int batchCount) {
  hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  hipDataType Atype = HIP_R_64F;
  hipDataType Btype = HIP_R_64F;
  hipDataType Ctype = HIP_R_64F;
  hipblasComputeType_t computeType = HIPBLAS_COMPUTE_64F;
  return hipblasGemmStridedBatchedEx(handle, transa, transb, m, n, k, alpha, A,
                                    Atype, lda, strideA, B, Btype, ldb, strideB,
                                    beta, C, Ctype, ldc, strideC, batchCount,
                                    computeType, algo);
}

hipblasStatus_t cublasHgemv2(hipblasHandle_t handle, hipblasOperation_t trans,
                            int m, int n, const __half *alpha, const __half *A,
                            int lda, const __half *x, int incx,
                            const __half *beta, __half *y, int incy) {
  float alpha_f = __half2float(*alpha);
  float beta_f = __half2float(*beta);
  return hipblasGemmEx(handle, trans, HIPBLAS_OP_N, m, 1, n, &alpha_f, A,
                      HIP_R_16F, m, x, HIP_R_16F, n, &beta_f, y, HIP_R_16F,
                      m, HIPBLAS_COMPUTE_32F_FAST_16F,
                      CUBLAS_GEMM_DFALT_TENSOR_OP);
}

hipblasStatus_t cublasHgemv3(hipblasHandle_t handle, hipblasOperation_t trans,
                            int m, int n, const __half *alpha, const __half *A,
                            int lda, const __half *x, int incx,
                            const __half *beta, __half *y, int incy) {
  return hipblasGemmEx(handle, trans, HIPBLAS_OP_N, m, 1, n, &alpha, A,
                      HIP_R_16F, m, x, HIP_R_16F, n, &beta, y, HIP_R_16F, m,
                      HIPBLAS_COMPUTE_16F, CUBLAS_GEMM_DFALT_TENSOR_OP);
}

hipblasStatus_t cublasHgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n, const __half *alpha, const __half *A,
                           int lda, const __half *x, int incx,
                           const __half *beta, __half *y, int incy) {
  return hipblasHgemm(handle, trans, HIPBLAS_OP_N, m, 1, n, alpha, A, lda, x, n,
                     beta, y, m);
}

hipblasStatus_t cublasHgemm2(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const __half *alpha, /* host or device pointer */
                            const __half *A, int lda, const __half *B, int ldb,
                            const __half *beta, /* host or device pointer */
                            __half *C, int ldc) {
  float alpha_f = __half2float(*alpha);
  float beta_f = __half2float(*beta);
  return hipblasGemmEx(handle, transa, transb, m, n, k, &alpha_f, A, HIP_R_16F,
                      lda, B, HIP_R_16F, ldb, &beta_f, C, HIP_R_16F, ldc,
                      HIPBLAS_COMPUTE_32F_FAST_16F, // Compute type
                      CUBLAS_GEMM_DFALT_TENSOR_OP  // Use Tensor Cores
  );
}

hipblasStatus_t cublasHgemm3(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const __half *alpha, /* host or device pointer */
                            const __half *A, int lda, const __half *B, int ldb,
                            const __half *beta, /* host or device pointer */
                            __half *C, int ldc) {
  return hipblasGemmEx(handle, transa, transb, m, n, k, &alpha, A, HIP_R_16F,
                      lda, B, HIP_R_16F, ldb, &beta, C, HIP_R_16F, ldc,
                      HIPBLAS_COMPUTE_16F,         // Compute type
                      CUBLAS_GEMM_DFALT_TENSOR_OP // Use Tensor Cores
  );
}

template <typename T> __global__ void initKernel(T *data, int size, T value) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < size) {
    data[idx] = value;
  }
}

void initVector_d(hipStream_t stream, double *data, int size, double value) {
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / blockSize;
  initKernel<<<numBlocks, blockSize, 0, stream>>>(data, size, value);
}

void initVector_f(hipStream_t stream, float *data, int size, float value) {
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / blockSize;
  initKernel<<<numBlocks, blockSize, 0, stream>>>(data, size, value);
}

void initVector_h(hipStream_t stream, __half *data, int size, __half value) {
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / blockSize;
  initKernel<<<numBlocks, blockSize, 0, stream>>>(data, size, value);
}

#if CUDART_VERSION >= 11020
#include <hip/hip_bf16.h>

void initVector_b(hipStream_t stream, __hip_bfloat16 *data, int size,
                  __hip_bfloat16 value) {
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / blockSize;
  initKernel<<<numBlocks, blockSize, 0, stream>>>(data, size, value);
}

hipblasStatus_t cublasBgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n, const __hip_bfloat16 *alpha,
                           const __hip_bfloat16 *A, int lda,
                           const __hip_bfloat16 *x, int incx,
                           const __hip_bfloat16 *beta, __hip_bfloat16 *y,
                           int incy) {
  float alpha_f = __bfloat162float(*alpha);
  float beta_f = __bfloat162float(*beta);
  return hipblasGemmEx(handle, trans, HIPBLAS_OP_N, m, 1, n, &alpha_f, A,
                      HIP_R_16BF, m, x, HIP_R_16BF, n, &beta_f, y,
                      HIP_R_16BF, m, HIPBLAS_COMPUTE_32F_FAST_16F,
                      CUBLAS_GEMM_DFALT_TENSOR_OP);
}

hipblasStatus_t
cublasBgemm(hipblasHandle_t handle, hipblasOperation_t transa,
            hipblasOperation_t transb, int m, int n, int k,
            const __hip_bfloat16 *alpha, /* host or device pointer */
            const __hip_bfloat16 *A, int lda, const __hip_bfloat16 *B, int ldb,
            const __hip_bfloat16 *beta, /* host or device pointer */
            __hip_bfloat16 *C, int ldc) {
  float alpha_f = __bfloat162float(*alpha);
  float beta_f = __bfloat162float(*beta);
  return hipblasGemmEx(handle, transa, transb, m, n, k, &alpha_f, A, HIP_R_16BF,
                      lda, B, HIP_R_16BF, ldb, &beta_f, C, HIP_R_16BF, ldc,
                      HIPBLAS_COMPUTE_32F_FAST_16F, // Compute type
                      CUBLAS_GEMM_DFALT_TENSOR_OP  // Use Tensor Cores
  );
}

hipblasStatus_t cublasBgemmsb(hipblasHandle_t handle, hipblasOperation_t transa,
                             hipblasOperation_t transb, int m, int n, int k,
                             const __hip_bfloat16 *alpha, const __hip_bfloat16 *A,
                             int lda, long long int strideA,
                             const __hip_bfloat16 *B, int ldb,
                             long long int strideB, const __hip_bfloat16 *beta,
                             __hip_bfloat16 *C, int ldc, long long int strideC,
                             int batchCount) {
  hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  hipDataType Atype = HIP_R_16BF;
  hipDataType Btype = HIP_R_16BF;
  hipDataType Ctype = HIP_R_16BF;
  hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32F;
  float alpha_f = __bfloat162float(*(__hip_bfloat16 *)alpha);
  float beta_f = __bfloat162float(*(__hip_bfloat16 *)beta);

  return hipblasGemmStridedBatchedEx(handle, transa, transb, m, n, k, &alpha_f,
                                    A, Atype, lda, strideA, B, Btype, ldb,
                                    strideB, &beta_f, C, Ctype, ldc, strideC,
                                    batchCount, computeType, algo);
}

#endif
